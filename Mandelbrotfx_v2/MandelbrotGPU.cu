
#include <hip/hip_runtime.h>
__device__ int returnNumberOfIterations(int n, double x, double y) {
    double zIM, zRE, tmp;
    zIM = zRE = 0;
    for(int i=0; i<n; i++) {
        if (sqrt(zRE * zRE + zIM * zIM) >= 2) {
            return i;
        }
        tmp = zRE * zRE - zIM * zIM + x;
        zIM = zIM * zRE + zIM * zRE + y;
        zRE = tmp;
    }
    return n;
}
extern "C"
__global__ void mandelbrot(int *tab, int *ints, double *doubles) {

    int height = ints[0];
    int width = ints[1];

    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= height * width) return;

    int N = ints[2];

    double minX = doubles[0];
    double maxX = doubles[1];
    double minY = doubles[2];
    double maxY = doubles[3];
    double  x,y;

    int row = i / width;
    int col = i % width;

    x = ((maxX - minX) * row) / height + minX;
    y = ((maxY - minY) * col) / width + minY;

    tab[i] = returnNumberOfIterations(N,x,y);
}